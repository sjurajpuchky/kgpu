#include "hip/hip_runtime.h"
/* ========================================================================== */
/*                                                                            */
/*   gramdiskfn.cu                                                            */
/*   (c) 2015 Juraj Puchky - Devtech                                          */
/*                                                                            */
/*   Common CUDA functions for GPU Ramdisk                                    */
/*                                                                            */
/* ========================================================================== */


#include "gramdisk.h"
#include <hip/hip_runtime.h>

  
static void gramdisk_request(struct request_queue *rq) 
{
    void *addr;
    struct request *req;
    int   len;
	
    while ((req = elv_next_request(rq)) != NULL) {
     if (blk_fs_request(req)) {  
     addr = gramdisk_data + (req->sector << 9);
	   len = req->current_nr_sectors << 9;
       switch(rq_data_dir(req)) {
        case WRITE:
         if(hipMemcpy(addr,req->buffer,len,hipMemcpyHostToDevice) == hipSuccess) {
         } else {
           printk("Cannot write a sector");
         }
        break;
        case READ:
         if(hipMemcpy(req->buffer,addr,len,hipMemcpyDeviceToHost) == hipSuccess) {
         } else {
           printk("Cannot read a sector");
         }
        break;
        default:
          printk("Unsupported command");
      }
     }
    }
}

// Init module
static int __init gramdisk_init(void)
{

  if(hipMalloc(&gramdisk_data, gramdisk_size) == hipSuccess) 
  {
    gramdisk_rq = blk_init_queue ( gramdisk_request, &gramdisk_lock);
    // Register block device major number
    if(register_blkdev(GRAMDISK_MAJOR_NUMBER,GRAMDISK_DEVICE_NAME)<=0) 
    {
       printk("Cannot register block device");
    } else {
       // Configure disk, set max sectors
       blk_queue_max_sectors (gramdisk_rq, gramdisk_sectors);
       // Configure disk, set size of sector
       blk_queue_hardsect_size (gramdisk_rq, gramdisk_size);
       // Alocate disk, set number of minor
       gramdisk_device = alloc_disk(GRAMDISK_MINOR_NUMBER);
       // Setup disk
       set_capacity(gramdisk_device, gramdisk_sectors*gramdisk_size);
       gramdisk_device->queue  = gramdisk_rq;
       gramdisk_device->major  = GRAMDISK_MAJOR_NUMBER;
       gramdisk_device->first_minor = 0;
       sprintf(gramdisk_device->disk_name, GRAMDISK_DEVICE_NAME);

        // Add the gendisk
        add_disk(gramdisk_device);       
    }     
  } else {
    printk("Cannot alloc memory on GPU");
  }
}

// Exit module
static void __exit gramdisk_exit(void)
{
    if(hipFree(&gramdisk_data) == hipSuccess) {
      blk_cleanup_queue(gramdisk_rq); 
    } else {
      printk("Cannot release memory on GPU");      
      
    }
}